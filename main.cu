#include "hip/hip_runtime.h"
//Time-stamp: <2013-11-28 11:35:34 hamada>
// FMA performance metor by Tsuyoshi Hamada

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

//#define REAL float
#define REAL double

#define DEVICE_ID (0)

#define NUM_MP (15)  // MultiProcessor (G92-GTS = 16, GT280 = 30)
#define NUM_SP_PER_BLOCK (192)
#define NUM_THREADS_PER_BLOCK (NUM_SP_PER_BLOCK*5)
#define NUM_BLOCKS (NUM_MP * 16)
#define NUM_ITERATIONS (1<<10)


/*
#define NUM_MP (16)
#define NUM_THREADS_PER_SM (384)
#define NUM_THREADS_PER_BLOCK (192)
#define NUM_BLOCKS ((NUM_THREADS_PER_SM / NUM_THREADS_PER_BLOCK) * NUM_MP)
#define NUM_ITERATIONS 30
*/

/*
#define NUM_MP (30)  // MultiProcessor (G92-GTS = 16, GT280 = 30)
#define NUM_THREADS_PER_BLOCK (128)
#define NUM_THREADS_PER_SM (128*16)
#define NUM_BLOCKS ((NUM_THREADS_PER_SM / NUM_THREADS_PER_BLOCK) * NUM_MP)
#define NUM_ITERATIONS (2048)
*/

/*
#define NUM_MP (30)  // MultiProcessor (G92-GTS = 16, GT280 = 30)
#define NUM_THREADS_PER_BLOCK (128)
#define NUM_THREADS_PER_SM (128*16)
#define NUM_BLOCKS ((NUM_THREADS_PER_SM / NUM_THREADS_PER_BLOCK) * NUM_MP)
#define NUM_ITERATIONS (2048)
*/


// 128 MAD instructions
#define FMAD128(a, b) \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \
  a *= b * a + b; \
  b *= a * b + a; \


__shared__ REAL result[NUM_THREADS_PER_BLOCK];


__device__ void fma128x16(REAL& a, REAL& b)
{
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
    FMAD128(a, b);
}

__global__ void gpu_func()
{
  REAL a = result[threadIdx.x];  // this ensures the mads don't get compiled out
  REAL b = 1.01f;

  for(int i=0; i<NUM_ITERATIONS; i++){
    fma128x16(a, b); // 1
    fma128x16(a, b); // 2
    fma128x16(a, b); // 3
    fma128x16(a, b); // 4
    fma128x16(a, b); // 5
    fma128x16(a, b); // 6
    fma128x16(a, b); // 7
    fma128x16(a, b); // 8
    fma128x16(a, b); // 9
  }
  /*
  fma128x16(a, b); // 10
  fma128x16(a, b); // 11
  fma128x16(a, b); // 12
  fma128x16(a, b); // 13
  fma128x16(a, b); // 14
  fma128x16(a, b); // 15
  fma128x16(a, b); // 16
  */
  result[threadIdx.x] = a + b;
}

#include <sys/time.h> 
#include <sys/resource.h>

double get_time(void)
{
  static struct timeval tv;
  static struct timezone tz;
  gettimeofday(&tv, &tz);
  return ((double)(tv.tv_sec  + tv.tv_usec*1.0e-6)); 
}


int run(int devid) 
{
	int n_dev = -1;
  hipGetDeviceCount(&n_dev);
	assert(0 < n_dev);
	printf("# of devices: %d\n", n_dev);

	// set Device ID as Round-robin
	devid = devid % n_dev;

	hipSetDevice(devid);
	hipGetDevice(&devid);
	printf("devid: %d\n", devid);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, devid);
	printf("[%d]: Device PCI Bus ID / PCI location ID: %x / %x\n",
				 devid, deviceProp.pciBusID, deviceProp.pciDeviceID);
	hipDeviceSynchronize();

  // execute kernel
  double time = get_time();
  gpu_func<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>();

  hipDeviceSynchronize();
  time = get_time() - time;

  // output results
  printf( "[%d]: Time: %f (ms)\n", devid, time*1000.0);
  const double num_fma = 16. * 9. * (double)NUM_ITERATIONS;
  const double flop = (double)(64. * 3. * num_fma) * (double)(NUM_BLOCKS * NUM_THREADS_PER_BLOCK);
  const double flops = flop/time;
	//  printf("[%d]: flop: %e\n", devid, flop);
  printf("[%d]: Gflops: %f\n", devid, flops / 1.0e+9 );
	printf("\n");
  return (0);
}

int main(int argc, char** argv) 
{
	int devid = DEVICE_ID;
	printf("N_BLOCK: %d\n", NUM_BLOCKS);
	printf("N_THREAD: %d\n", NUM_THREADS_PER_BLOCK);

	if(argc > 1){
		devid = atoi(argv[1]);
		run(devid);
	}else{
		for(devid = 0; devid < 16; devid++)	run(devid);
	}
	return 0;
}
